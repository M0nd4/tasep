#include "hip/hip_runtime.h"
#include "cu_lattice.hpp"

#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <thrust/device_vector.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iomanip>

struct Ribosome {
    double time;
    int pos;
};


struct Codon {
    double time;
    double rate;
    bool occupied;
    double accumtime;
};


__global__ static 
void setupRand ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
} 


__device__ inline static
double sampleTime (double rate, hiprandState* globalState)
{
    hiprandState localState = globalState[threadIdx.x];
    double u = hiprand_uniform_double( &localState );
    globalState[threadIdx.x] = localState;
    if (u == 0) u += 0.1; // should almost never happen
    return -log(u) / rate;  // let rate == 0 throw
}


//__host__ __device__ inline static
__global__ static 
void updatePolysome (Codon* codons, Ribosome* ribosomes, int length, hiprandState* globalState )
{
    // copy ribosom data from global memory to registers
    int riboId = threadIdx.x;
    int nextId = (riboId + blockDim.x - 1) % blockDim.x;
    Ribosome ribo     = ribosomes[riboId];
    Ribosome riboNext = ribosomes[nextId];
    __syncthreads();

    // get the position of the next and previous ribosomes
    bool nextIsFar = (riboNext.pos - ribo.pos + length) % length > 1 || (ribo.pos != 0 && riboNext.pos == 0);
    if (!nextIsFar) return;

    // sample the period
    double dt = sampleTime(codons[ribo.pos].rate, globalState);

    // flip two codons and update the ribosome position
    int newpos = (ribo.pos + length + 1) % length;
    codons[newpos].occupied = true;
    codons[ribo.pos].occupied = false;
    ribosomes[riboId].pos = newpos;

    // wait for the following codon to clear (if necessary), then add dt
    double t = max(ribo.time, codons[newpos].time) + dt;
    // find the occupied time
    codons[ribo.pos].accumtime += (t - codons[ribo.pos].time);
    // update its own time and the time of the codon
    ribosomes[riboId].time = t;
    codons[ribo.pos].time = t;   // for the next ribosome
    codons[newpos].time = t;     // for computing time of occupancy by next ribosome

    // take care of the border
    if (ribo.pos == length - 1)
    {
        ribosomes[riboId].time = 0;
        codons[0].time = 0;
        codons[0].occupied = false;
    }
}

/*
__global__ static 
void computePolysome (double* rates, bool* cover, int length, int padding)
{
}
*/

using namespace std;


vector<double> runSinglePolysome (const vector<double>& rates, double initRate)
{
    int padding = 1;

    // pad the vector
    int length = rates.size();
    int lengthPadded = length + padding;

    cout << "length: " << length << endl;
    cout << "padding: " << padding << endl;

    // init codons
    thrust::device_vector<Codon> codonsVector (lengthPadded);
    for (int i = 0; i != length; ++i)
    {
        Codon codon; codon.rate = rates[i]; codon.time = 0; codon.occupied = false; codon.accumtime = 0;
        codonsVector[i+padding] = codon;
    }
    Codon codon0; codon0.rate = initRate; codon0.time = 0; codon0.occupied = false; codon0.accumtime = 0;
    codonsVector.front() = codon0;
    Codon codon1 = codonsVector[1];
    codon1.occupied = true;
    codonsVector[1] = codon1;
    Codon* deviceCodons = thrust::raw_pointer_cast( &codonsVector[0] );

    // init ribosomes
    //const int RiboWidth = 10;
    const int numRibosomes = 10;//((lengthPadded - 1) / 32 / RiboWidth + 1) * 32;
    cout << "numRibosomes: " << numRibosomes << endl;
    thrust::device_vector<Ribosome> ribosomesVector (numRibosomes);
    Ribosome ribo00; ribo00.pos = 0; ribo00.time = 0;
    Ribosome ribo10; ribo10.pos = 1; ribo10.time = 0;
    for (int i = 0; i != numRibosomes; ++i) ribosomesVector[i] = ribo00;
    ribosomesVector[0] = ribo10;
    Ribosome* deviceRibosomes = thrust::raw_pointer_cast( &ribosomesVector[0] );

    // setup seeds
    hiprandState* deviceStates;
    hipMalloc ( &deviceStates, numRibosomes * sizeof(hiprandState) );
    setupRand <<< 1, numRibosomes >>> ( deviceStates, time(NULL) );    
        
    //cout << "iteration   codon.occupied   codon.time   ribosome.pos   ribosome.time" << endl;
    cout << "iteration   codon.occupied   codon.accumtime" << endl;
    for (int it = 0; it != 30; ++it)
    {
        cout << "iteration " << it << ": " << flush;
        for (int i = 0; i != lengthPadded; ++i)
        {
            Codon codon = codonsVector[i];
            cout << (codon.occupied ? '*' : '.');
        }
        cout << "    ";
        /*
        {
            Codon codon = codonsVector[i];
            cout << setprecision(3) << codon.time << " ";
        }
        cout << "    ";
        *//*
        for (int i = 0; i != numRibosomes; ++i)
        {
            Ribosome ribosome = ribosomesVector[i];
            cout << ribosome.pos << " ";
        }
        cout << "    ";
        *//*
        for (int i = 0; i != numRibosomes; ++i)
        {
            Ribosome ribosome = ribosomesVector[i];
            cout << setprecision(3) << ribosome.time << " ";
        }
        */
        cout << "    ";
        for (int i = 0; i != lengthPadded; ++i)
        {
            Codon codon = codonsVector[i];
            cout << setprecision(3) << codon.accumtime << " ";
        }
        
        cout << endl;

        updatePolysome 
            <<< 1, numRibosomes, 0 >>> 
            (deviceCodons, deviceRibosomes, lengthPadded, deviceStates);

    }

    const int epoch = 1;
    vector<double> probs (length);
    for (int i = 0; i != probs.size(); ++i)
    {
        Codon codon = codonsVector[i+padding];
        probs[i] = codon.accumtime / epoch;
    }

    return probs;
}
