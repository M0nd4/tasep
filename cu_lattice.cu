#include "hip/hip_runtime.h"
#include "cu_lattice.hpp"

#include <stdio.h>
#include <stdlib.h>

// for one single polysom  only
__global__ static 
void computePolysome (double* deviceRates, int length)
{
	
}


void runSinglePolysome (double* rates, int length)
{
	double *deviceRates;
    hipMalloc(&deviceRates, length * sizeof(double));
    
    hipMemcpy(&deviceRates, rates,
               length * sizeof(double), hipMemcpyHostToDevice);

    
    computePolysome
        <<< 1, 1, 0 >>>
        (deviceRates, length);
}
