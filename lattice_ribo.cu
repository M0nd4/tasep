#include "hip/hip_runtime.h"
#include "lattice.hpp"

#include <list>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <algorithm>
#include <iomanip>
#include <sys/time.h>

#include <thrust/device_vector.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// FIXME: for many blocks probably need to create many states
__global__ static
void setupRand ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
} 


__device__ inline static
double sampleTime (double rate, hiprandState* globalState)
{
    hiprandState localState = globalState[threadIdx.x];
    double u = hiprand_uniform_double( &localState );
    globalState[threadIdx.x] = localState;
    if (u == 0) u += 0.1; // should almost never happen
    return -log(u) / rate;  // let rate == 0 throw
}


__device__ static
//__global__ static 
void updatePolysome (Codon* codons, Ribosome* ribosomes, int length, double epoch, hiprandState* globalState)
{
    // TODO: ribosomes AoS to SoA

    // copy ribosom data from global memory to registers
    int riboId = threadIdx.x;
    int nextId = (riboId + blockDim.x - 1) % blockDim.x;
    Ribosome ribo     = ribosomes[riboId];
    Ribosome nextribo = ribosomes[nextId];

    // most ribos are inactive. Check and return early
    if (ribo.pos == 0 && nextribo.pos == 0) return;

    int pos = ribo.pos;
    int nextpos = ((pos - RiboKeyCodon + RiboWidth) % length);
    nextpos *= int(nextpos > pos);  // zero if ribosome is about to jump off

    // copy codon data from global memory to registers
    Codon codon = codons[pos];
    Codon nextcodon = codons[nextpos];

    // after copy
    __syncthreads();

    // range of covered codons, the range follows the convention [a, b)
    int beginCoveredPos = max (pos - RiboKeyCodon, 1);    // do not change virtual codon 0
    int endCoveredPos   = min (pos - RiboKeyCodon + RiboWidth, length);
    
    // update current time with time of the next codon
    double t0 = max(codon.time, nextcodon.time);
    //if (nextpos == 0) t0 = codon.time;  // debugging, shouldn't make differrence
    ribosomes[riboId].time = t0;  // so that ribos, stalled after dead ribos get to epoch eventually
    codons[pos].accumtime += t0 - codon.time; 
    for (int i = beginCoveredPos; i < endCoveredPos; ++i) codons[i].time = t0;

    // do not jump if can not
    bool nextIsFar = (nextribo.pos - pos > RiboWidth) || (pos != 0 && nextribo.pos == 0); 
    if (!nextIsFar) return;

    // sample the period
    double dt = sampleTime(codon.rate, globalState);
    dt = min(dt, epoch - t0);     // when about to finish
    double t = t0 + dt;

    // update times of the ribo and of the codon
    codons[pos].accumtime += dt;
    ribosomes[riboId].time = t;
    for (int i = beginCoveredPos; i < endCoveredPos; ++i) codons[i].time = t;
    codons[nextpos].time = t;     // for computing time of occupancy by next ribosome

    // finish simulation for this ribo when time reaches the epoch
    if (t >= epoch) return;

    // flip two codons and update the ribosome position
    int jumppos = (pos + 1) % length;
    codons[jumppos].occupied = true;
    codons[pos].occupied = false;
    ribosomes[riboId].pos = jumppos;

    // zero when at the border
    ribosomes[riboId].time *= int(jumppos != 0);
}

__device__ static
int countActiveRibos (Ribosome* ribosomes, double epoch)
{
    // TODO: rewrite with reduce
    int countNonactive = 0;
    for (int i = 0; i != blockDim.x; ++i)
    {
        Ribosome ribo = ribosomes[i];
        if (ribo.time >= epoch || ribo.pos == 0)
            ++countNonactive;
    }
    return blockDim.x - countNonactive;
}

__device__ static
bool hasHitEpoch (Ribosome* ribosomes, double epoch)
{
    for (int i = 0; i != blockDim.x; ++i)
        if (ribosomes[i].time >= epoch) return true;
    return false;
}

// pass info forward and backwards
struct In {
    int maxIterMult;
    double epoch;
    int frontpadding;
    int iters4display;
};

struct Out {
    int iter, iterHitEpoch;
    double* prob;
    char* occupancy;
    int* activeRibos;
    double *codonTimes;
};


__global__ static 
void computePolysome (Codon** codonsPtr, Ribosome** ribosomesPtr, int* lengthsPtr, 
                      In* inPtr, Out* outPtr, hiprandState* globalState, int verbose = 0)
{
    __shared__ int activeRibos;
   
    Codon codon0; codon0.occupied = false; codon0.time = 0; codon0.accumtime = 0;

    // each block has its own arrays (numRibosomes is the same in every block)
    Codon*    codons = codonsPtr[blockIdx.x];
    Ribosome* ribosomes = ribosomesPtr[blockIdx.x];
    int       length = lengthsPtr[blockIdx.x];
    In        in = inPtr[blockIdx.x];
    Out       out = outPtr[blockIdx.x];

    for (out.iter = 0; out.iter != in.maxIterMult * length; ++out.iter)
    {
        // stop condition
        if (threadIdx.x == 0)
        {
            activeRibos = countActiveRibos (ribosomes, in.epoch);
            if (verbose > 2 && !out.iterHitEpoch && hasHitEpoch (ribosomes, in.epoch)) 
                out.iterHitEpoch = out.iter;
        }
        __syncthreads();
        if (activeRibos == 0) break;

        // write debug info into specially pre-allocated memory 
        if (verbose > 2 && out.iter < in.iters4display)
        {
            if (threadIdx.x == 0) out.activeRibos[out.iter] = activeRibos;
            for (int i = threadIdx.x; i < length; i += blockDim.x)
            {
                out.occupancy[out.iter * length + i] = codons[i].occupied;
                out.codonTimes[out.iter * length + i] = codons[i].time;  
            }
            __syncthreads();
        }

        updatePolysome (codons, ribosomes, length, in.epoch, globalState);
        __syncthreads();
        if (threadIdx.x == 0)
            codons[0] = codon0;
    }

    // calculate resulting probability
    __syncthreads();
    
    for (int i = threadIdx.x; i < length; i += blockDim.x)
        out.prob[i] = codons[i+in.frontpadding].accumtime / in.epoch;
    
    if (threadIdx.x == 0)
        outPtr[blockIdx.x] = out;
}



using namespace std;


Codon* initCodons (const vector<double>& rates)
{
    int length = rates.size();
    vector<Codon> hostCodons (length);
    for (int i = 0; i != length; ++i)
    {
        Codon codon; codon.rate = rates[i]; codon.time = 0; codon.occupied = (i == 1); codon.accumtime = 0;
        hostCodons[i] = codon;
    }
    Codon* deviceCodons;
    hipMalloc(&deviceCodons, length*sizeof(Codon));
    hipMemcpy(deviceCodons, &hostCodons[0], length*sizeof(Codon), hipMemcpyHostToDevice);
    return deviceCodons;
}

Ribosome* initRibosomes (int numRibosomes)
{
    vector<Ribosome> hostRibosomes (numRibosomes);
    Ribosome ribo00; ribo00.pos = 0; ribo00.time = 0;
    Ribosome ribo10; ribo10.pos = 1; ribo10.time = 0;
    for (int i = 0; i != numRibosomes; ++i) hostRibosomes[i] = ribo00;
    hostRibosomes[0] = ribo10;
    Ribosome* deviceRibosomes;
    hipMalloc(&deviceRibosomes, numRibosomes*sizeof(Ribosome));
    hipMemcpy(deviceRibosomes, &hostRibosomes[0], numRibosomes*sizeof(Ribosome), hipMemcpyHostToDevice);
    return deviceRibosomes;
}

void initDebug (In& in, Out& out, int length, int iters4display)
{
    int space4occupancy = iters4display*length*sizeof(char);
    int space4time = in.iters4display*length*sizeof(double);
    char* deviceOccupancy;
    int* deviceActiveRibos;
    double* deviceCodonTimes;
    if (iters4display) 
    {
        hipMalloc (&deviceOccupancy, space4occupancy);
        hipMemset (deviceOccupancy, 0, space4occupancy);
        hipMalloc (&deviceActiveRibos, iters4display*sizeof(int));
        hipMemset (deviceActiveRibos, 0, iters4display*sizeof(int));
        hipMalloc (&deviceCodonTimes, space4time);
        hipMemset (deviceCodonTimes, 0, space4time);
    }
    in.iters4display = iters4display;
    out.occupancy = deviceOccupancy;
    out.activeRibos = deviceActiveRibos;
    out.codonTimes = deviceCodonTimes;
    out.iterHitEpoch = 0;
}

void printDebug (const In& in, const Out& out, int length)
{
    if (in.iters4display)
    {
        int space4occupancy = in.iters4display*length*sizeof(char);
        int space4time = in.iters4display*length*sizeof(double);
        vector<char> vectorOccupancy (in.iters4display*length);
        hipMemcpy (&vectorOccupancy[0], out.occupancy, space4occupancy, hipMemcpyDeviceToHost);
        vector<int> vectorActiveRibos (in.iters4display);
        hipMemcpy (&vectorActiveRibos[0], out.activeRibos, in.iters4display*sizeof(int), hipMemcpyDeviceToHost);
        vector<double> vectorCodonTimes (in.iters4display*length);
        hipMemcpy (&vectorCodonTimes[0], out.codonTimes, space4time, hipMemcpyDeviceToHost);
        hipFree (out.occupancy);
        hipFree (out.activeRibos);
        hipFree (out.codonTimes);
        for (int iter = 0; iter != min(in.iters4display, out.iter); ++iter)
        {
            cout << (out.iterHitEpoch == iter ? "/" : " ");
            cout << setw(3) << iter << "  &  " << setw(3) << vectorActiveRibos[iter] << "  &  ";
            for (int i = 0; i != length; ++i)
                cout << (vectorOccupancy[iter * length + i] ? '*' : '.');
            cout << "  ";
            for (int i = 0; i != length; ++i)
                cout << setprecision(3) << setw(5) << vectorCodonTimes[iter * length + i] << " ";
            cout << endl;
        }
    }
}



void runSinglePolysome (const vector<double>& rates, double epoch, 
                        vector<double>& probs, int verbose)
{ 
    int length = rates.size();
    if (verbose > 1) cout << "length: " << length << endl;
    
    // init
    const int numRibosomes = min (1024, ((length - 1) / 32 / RiboWidth + 1) * 32);
    Ribosome* deviceRibosomes = initRibosomes(numRibosomes);
    Codon*    deviceCodons    = initCodons(rates);

    // set up seeds
    hiprandState* deviceStates;
    hipMalloc ( &deviceStates, numRibosomes * sizeof(hiprandState) );
    setupRand <<< 1, numRibosomes >>> ( deviceStates, time(NULL) );

    // pass constants
    In in; in.epoch = epoch; in.maxIterMult = MaxIterMult; in.frontpadding = 1;
    if (verbose > 1)
        cout << "epoch: " << epoch << ", numRibos: " << numRibosomes << endl;

    // info to return
    double* deviceProb;
    hipMalloc (&deviceProb, length*sizeof(double));
    Out out; out.prob = deviceProb;

    // debugging/visualization info to return
    if (verbose > 2) initDebug (in, out, length, 200);

    // copy the in/out structs to device
    thrust::device_vector<Codon*> codonsPtr       (1, deviceCodons);
    thrust::device_vector<Ribosome*> ribosomesPtr (1, deviceRibosomes);
    thrust::device_vector<int> lengthPtr          (1, length);
    thrust::device_vector<In> inPtr               (1, in);
    thrust::device_vector<Out> outPtr             (1, out);

    if (verbose > 1) cout << "in: " << in.epoch << " " << in.maxIterMult << endl;

    computePolysome <<< 1, numRibosomes >>> (thrust::raw_pointer_cast( codonsPtr.data() ), 
                                             thrust::raw_pointer_cast( ribosomesPtr.data() ), 
                                             thrust::raw_pointer_cast( lengthPtr.data() ), 
                                             thrust::raw_pointer_cast( inPtr.data() ),
                                             thrust::raw_pointer_cast( outPtr.data() ),
                                             deviceStates, verbose);

    out = outPtr[0];
    if (verbose > 1)
        cout << "finished in " << out.iter << " iterations" << endl;
    if (out.iter >= in.maxIterMult * length)
        cerr << "warning: reached the maximum number of iterations" << endl;

    // debugging/visualization info
    if (verbose > 2) printDebug (in, out, length);

    // write result
    probs.resize(length);
    hipMemcpy (&probs[0], deviceProb, length*sizeof(double), hipMemcpyDeviceToHost);

    // clean up
    hipFree (deviceProb);
    hipFree (deviceCodons);
    hipFree (deviceRibosomes);
    hipFree (deviceStates);
}



// sort input array and return permutation indices. Can be done with lambdas in Cuda 7.
// TODO: do sort in a kernel
struct LengthComparatorByIndex
{
    LengthComparatorByIndex (const vector<vector<double> >& data) : m_data(data) { }
    bool operator()(int left, int right) const { return m_data[left].size() < m_data[right].size(); }
    const vector< vector<double> > & m_data;
};
vector<size_t> orderedLength (const vector< vector<double> >& values) {
    vector<size_t> indices (values.size());
    for (int i = 0; i != values.size(); ++i) indices[i] = i;
    sort( indices.begin(), indices.end(), LengthComparatorByIndex(values));
    return indices;
}


void runMultiplePolysomes (const vector< vector<double> > rates, double epoch,
                           vector< vector<double> >& probs, int verbose)
{
    // time profile
    struct timeval tv1, tv2;
    gettimeofday(&tv1, NULL);
    hipFree(0);
    gettimeofday(&tv2, NULL);
    /*printf("CUDA init time, sec: %f\n",
           (double) (tv2.tv_usec - tv1.tv_usec) / (double)1000000 +
           (double) (tv2.tv_sec - tv1.tv_sec));
    */gettimeofday(&tv1, NULL);

    int numRNAs = rates.size();
    probs.resize(numRNAs);

    const int MinBlockPerSplit = 70;
    const double SplitReductionFactor = 1.5;
    // sort rates vectors based on length
    vector<size_t> indices = orderedLength (rates);
    // form a list of indices where RNAs will be split
    vector<int> indicesOfSplit (1, indices.size()-1);
    // split by length. Each time length is halved, it is a split
    for (int i = indices.size()-1; i != -1; --i)
    {
        // stop condition
        if (rates[indices[i]].size() < 32 * RiboWidth) break;
        // go until have at least 32 RNAs
        if (indicesOfSplit.back() - i < MinBlockPerSplit) continue;
        // point where one more split is done
        if (rates[indices[i]].size() < rates[indices[indicesOfSplit.back()]].size() / SplitReductionFactor)
            indicesOfSplit.push_back(i);
    }
    reverse (indicesOfSplit.begin(), indicesOfSplit.end());

    for (int split = 0; split != indicesOfSplit.size(); ++split)
    {
        int splitSize = indicesOfSplit[split] - (split == 0 ? -1 : indicesOfSplit[split-1]);
        int maxLength = rates[indices[indicesOfSplit[split]]].size();
        int numRibosomes = min (1024, ((maxLength / RiboWidth - 1) / 32 + 1) * 32);
        if (verbose) 
            cout << "splitSize: " << setw(4) << splitSize 
                 << ", maxLength: " << setw(4) << maxLength 
                 << ", numRibos: " << setw(5) << numRibosomes << endl;

        // set up seeds
        hiprandState* deviceStates;
        hipMalloc ( &deviceStates, numRibosomes * sizeof(hiprandState) );
        setupRand <<< 1, numRibosomes >>> ( deviceStates, time(NULL) );

        // one element per RNA
        thrust::device_vector<Codon*> codonsPtr       (splitSize);
        thrust::device_vector<Ribosome*> ribosomesPtr (splitSize);
        thrust::device_vector<int> lengthPtr          (splitSize);
        thrust::device_vector<In> inPtr               (splitSize);
        thrust::device_vector<Out> outPtr             (splitSize);

        // prepare inputs
        int beginIndex = (split == 0 ? 0 : indicesOfSplit[split-1] + 1);
        int endIndex = indicesOfSplit[split] + 1;
        int index, rnaId;
        for (index = beginIndex, rnaId = 0; index != endIndex; ++index, ++rnaId)
        {
            int rna = indices[index];
            int length = rates[rna].size();

            // init
            Ribosome* deviceRibosomes = initRibosomes(numRibosomes);
            Codon*    deviceCodons    = initCodons(rates[rna]);

            // pass constants
            In in; in.epoch = epoch; in.maxIterMult = MaxIterMult; in.frontpadding = 1;

            // info to return
            double* deviceProb;
            hipMalloc (&deviceProb, length*sizeof(double));
            Out out; out.prob = deviceProb;

            // debugging/visualization info to return
            if (verbose > 2) initDebug (in, out, length, 200);

            // copy the in/out structs to device
            codonsPtr[rnaId]    = deviceCodons;
            ribosomesPtr[rnaId] = deviceRibosomes;
            lengthPtr[rnaId]    = length;
            inPtr[rnaId]        = in;
            outPtr[rnaId]       = out;
        }

        computePolysome <<< splitSize, numRibosomes >>> (thrust::raw_pointer_cast( codonsPtr.data() ), 
                                                       thrust::raw_pointer_cast( ribosomesPtr.data() ), 
                                                       thrust::raw_pointer_cast( lengthPtr.data() ), 
                                                       thrust::raw_pointer_cast( inPtr.data() ),
                                                       thrust::raw_pointer_cast( outPtr.data() ),
                                                       deviceStates, verbose);

        // process outputs
        for (index = beginIndex, rnaId = 0; index != endIndex; ++index, ++rnaId)
        {
            int rna = indices[index];
            int length = rates[rna].size();

            Codon* deviceCodons       = codonsPtr[rnaId];
            Ribosome* deviceRibosomes = ribosomesPtr[rnaId];
            Out out                   = outPtr[rnaId];
            In in                     = inPtr[rnaId];
            double* deviceProb        = out.prob;

            int numRibosomes = min (1024, ((maxLength / RiboWidth - 1) / 32 + 1) * 32);
            if (verbose > 1)
                cout << "rna: " << setw(4) << rna 
                     << ", length: " << setw(4) << length 
                     << ", numRibos: " << setw(4) << numRibosomes
                     << ", finished in " << out.iter << " iterations" << endl;
            if (out.iter >= in.maxIterMult * length)
                cerr << "warning: rna " << rna << " reached the maximum number of iterations" << endl;

            // debugging/visualization info
            if (verbose > 2) printDebug (in, out, length);

            // write result
            probs[rna].resize(length);
            hipMemcpy (&probs[rna][0], deviceProb, length*sizeof(double), hipMemcpyDeviceToHost);

            // clean up
            hipFree (deviceProb); 
            hipFree (deviceCodons);
            hipFree (deviceRibosomes);
        }

        hipFree (deviceStates);
    } // split

    gettimeofday(&tv2, NULL);
    printf("CUDA-main-time-sec: %f\n",
           (double) (tv2.tv_usec - tv1.tv_usec) / (double)1000000 +
           (double) (tv2.tv_sec - tv1.tv_sec));
}
