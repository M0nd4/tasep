#include "hip/hip_runtime.h"
#include "lattice.hpp"

#include <list>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <algorithm>

#include <thrust/device_vector.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iomanip>

#include <sys/time.h>

// FIXME: for many blocks probably need to create many states
__global__ static
void setupRand ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
} 


__device__ inline static
double sampleTime (double rate, hiprandState* globalState)
{
    hiprandState localState = globalState[threadIdx.x];
    double u = hiprand_uniform_double( &localState );
    globalState[threadIdx.x] = localState;
    if (u == 0) u += 0.1; // should almost never happen
    return -log(u) / rate;  // let rate == 0 throw
}


__device__ static
//__global__ static 
void updatePolysome (Codon* codons, Ribosome* ribosomes, int length, double epoch, hiprandState* globalState)
{
    // TODO: ribosomes AoS to SoA

    // copy ribosom data from global memory to registers
    int riboId = threadIdx.x;
    int nextId = (riboId + blockDim.x - 1) % blockDim.x;
    Ribosome ribo     = ribosomes[riboId];
    Ribosome nextribo = ribosomes[nextId];

    // most ribos are inactive. Check and return early
    if (ribo.pos == 0 && nextribo.pos == 0) return;

    int pos = ribo.pos;
    int nextpos = (pos + length + (RiboWidth - RiboKeyCodon)) % length;
    
    // copy codon data from global memory to registers
    Codon codon = codons[pos];
    Codon nextcodon = codons[nextpos];

    // after copy
    __syncthreads();

    // range of covered codons, the range follows the convention [a, b)
    int beginCoveredPos = max (pos - RiboKeyCodon, 0);
    int endCoveredPos   = min (pos - RiboKeyCodon + RiboWidth, length);
    
    // update current time with time of the next codon
    double t0 = max(codon.time, nextcodon.time);
    ribosomes[riboId].time = t0;
    codons[pos].accumtime += t0 - codon.time; 
    for (int i = beginCoveredPos; i != endCoveredPos; ++i) codons[i].time = t0;

    // do not jump if can not
    bool nextIsFar = (nextribo.pos - pos > RiboWidth) || (pos != 0 && nextribo.pos == 0); 
    if (!nextIsFar) return;

    // sample the period
    double dt = sampleTime(codon.rate, globalState);
    dt = min(dt, epoch - t0);     // when about to finish
    double t = t0 + dt;

    // update times of the ribo and of the codon
    codons[pos].accumtime += dt;
    ribosomes[riboId].time = t;
    for (int i = beginCoveredPos; i != endCoveredPos; ++i) codons[i].time = t;
    codons[nextpos].time = t;     // for computing time of occupancy by next ribosome

    // finish simulation for this ribo when time reaches the epoch
    if (t >= epoch) return;

    // flip two codons and update the ribosome position
    int jumppos = (pos + 1) % length;
    codons[jumppos].occupied = true;
    codons[pos].occupied = false;
    ribosomes[riboId].pos = jumppos;

    // zero when at the border
    ribosomes[riboId].time *= (pos == length - 1);

    __syncthreads();
    if (threadIdx.x == 0)
    {
        codons[0].time = 0;
        codons[0].occupied = false;
    }
}


__device__ static
int countActiveRibos (Ribosome* ribosomes, double epoch)
{
    // TODO: rewrite with reduce
    int countNonactive = 0;
    for (int i = 0; i != blockDim.x; ++i)
    {
        Ribosome ribo = ribosomes[i];
        if (ribo.time >= epoch || ribo.pos == 0)
            ++countNonactive;
    }
    return blockDim.x - countNonactive;
}


// pass info forward and backwards
struct In {
    int maxIterMult;
    double epoch;
    int frontpadding;
    int iters4display;
};

struct Out {
    int iter;
    double* prob;
    char* occupancy;
    int* activeRibos;
};


__global__ static 
void computePolysome (Codon** codonsPtr, Ribosome** ribosomesPtr, int* lengthsPtr, 
                      In* inPtr, Out* outPtr, hiprandState* globalState, int verbose = 0)
{
    __shared__ int activeRibos;
   
    // each block has its own arrays (numRibosomes is the same in every block)
    Codon*    codons = codonsPtr[blockIdx.x];
    Ribosome* ribosomes = ribosomesPtr[blockIdx.x];
    int       length = lengthsPtr[blockIdx.x];
    In        in = inPtr[blockIdx.x];
    Out       out = outPtr[blockIdx.x];

    for (out.iter = 0; out.iter != in.maxIterMult * length; ++out.iter)
    {
        // stop condition
        if (threadIdx.x == 0)
            activeRibos = countActiveRibos (ribosomes, in.epoch);
        __syncthreads();
        if (activeRibos == 0) break;

        // write occupancy into specially pre-allocated memory 
        if (verbose > 1 && out.iter < in.iters4display)
        {
            if (threadIdx.x == 0) out.activeRibos[out.iter] = activeRibos;
            for (int i = threadIdx.x; i < length; i += blockDim.x)
                out.occupancy[out.iter * length + i] = codons[i].occupied;
            __syncthreads();
        }

        updatePolysome (codons, ribosomes, length, in.epoch, globalState);
    }

    // calculate resulting probability
    __syncthreads();
    
    for (int i = threadIdx.x; i < length; i += blockDim.x)
        out.prob[i] = codons[i+in.frontpadding].accumtime / in.epoch;
    
    if (threadIdx.x == 0)
        outPtr[blockIdx.x] = out;
}



using namespace std;


Codon* initCodons (const vector<double>& rates)
{
    int length = rates.size();
    vector<Codon> hostCodons (length);
    for (int i = 0; i != length; ++i)
    {
        Codon codon; codon.rate = rates[i]; codon.time = 0; codon.occupied = (i == 1); codon.accumtime = 0;
        hostCodons[i] = codon;
    }
    Codon* deviceCodons;
    hipMalloc(&deviceCodons, length*sizeof(Codon));
    hipMemcpy(deviceCodons, &hostCodons[0], length*sizeof(Codon), hipMemcpyHostToDevice);
    return deviceCodons;
}

Ribosome* initRibosomes (int numRibosomes)
{
    vector<Ribosome> hostRibosomes (numRibosomes);
    Ribosome ribo00; ribo00.pos = 0; ribo00.time = 0;
    Ribosome ribo10; ribo10.pos = 1; ribo10.time = 0;
    for (int i = 0; i != numRibosomes; ++i) hostRibosomes[i] = ribo00;
    hostRibosomes[0] = ribo10;
    Ribosome* deviceRibosomes;
    hipMalloc(&deviceRibosomes, numRibosomes*sizeof(Ribosome));
    hipMemcpy(deviceRibosomes, &hostRibosomes[0], numRibosomes*sizeof(Ribosome), hipMemcpyHostToDevice);
    return deviceRibosomes;
}

void initDebug (In& in, Out& out, int length, int iters4display)
{
    int space4occupancy = iters4display*length*sizeof(char);
    char* deviceOccupancy;
    int* deviceActiveRibos;
    if (iters4display) 
    {
        hipMalloc (&deviceOccupancy, space4occupancy);
        hipMemset (deviceOccupancy, 0, space4occupancy);
        hipMalloc (&deviceActiveRibos, iters4display*sizeof(int));
        hipMemset (deviceActiveRibos, 0, iters4display*sizeof(int));
    }
    in.iters4display = iters4display;
    out.occupancy = deviceOccupancy;
    out.activeRibos = deviceActiveRibos;                        
}

void printDebug (const In& in, const Out& out, int length)
{
    if (in.iters4display)
    {
        int space4occupancy = in.iters4display*length*sizeof(char);
        vector<char> vectorOccupancy (in.iters4display*length);
        hipMemcpy (&vectorOccupancy[0], out.occupancy, space4occupancy, hipMemcpyDeviceToHost);
        vector<int> vectorActiveRibos (in.iters4display);
        hipMemcpy (&vectorActiveRibos[0], out.activeRibos, in.iters4display*sizeof(int), hipMemcpyDeviceToHost);
        for (int iter = 0; iter != min(in.iters4display, out.iter); ++iter)
        {
            cout << setw(3) << iter << "  &  " << setw(3) << vectorActiveRibos[iter] << "  &  ";
            for (int i = 0; i != length; ++i)
                cout << (vectorOccupancy[iter * length + i] ? '*' : '.');
            cout << endl;
        }
        hipFree (out.occupancy);
        hipFree (out.activeRibos);
    }
}



void runSinglePolysome (const vector<double>& rates, double epoch, 
                        vector<double>& probs, int verbose)
{
    struct timeval tv1, tv2;
    
    gettimeofday(&tv1, NULL);
    hipFree(0);
    gettimeofday(&tv2, NULL);
    printf("CUDA init: Time taken in execution = %f seconds\n",
           (double) (tv2.tv_usec - tv1.tv_usec) / (double)1000000 +
           (double) (tv2.tv_sec - tv1.tv_sec));
           
    
    gettimeofday(&tv1, NULL); 
    int length = rates.size();
    cout << "length: " << length << endl;
    
    // init
    const int numRibosomes = min (1024, ((length - 1) / 32 / RiboWidth + 1) * 32);
    Ribosome* deviceRibosomes = initRibosomes(numRibosomes);
    Codon*    deviceCodons    = initCodons(rates);

    // set up seeds
    hiprandState* deviceStates;
    hipMalloc ( &deviceStates, numRibosomes * sizeof(hiprandState) );
    setupRand <<< 1, numRibosomes >>> ( deviceStates, time(NULL) );

    // pass constants
    In in; in.epoch = epoch; in.maxIterMult = 100; in.frontpadding = 1;
    cout << "epoch: " << epoch << ", MaxIterMult: " << in.maxIterMult << ", numRibos: " << numRibosomes << endl;

    // info to return
    double* deviceProb;
    hipMalloc (&deviceProb, length*sizeof(double));
    Out out; out.prob = deviceProb;

    // debugging/visualization info to return
    if (verbose) initDebug (in, out, length, (verbose > 1 ? 200 : 0));

    // copy the in/out structs to device
    thrust::device_vector<Codon*> codonsPtr       (1, deviceCodons);
    thrust::device_vector<Ribosome*> ribosomesPtr (1, deviceRibosomes);
    thrust::device_vector<int> lengthPtr          (1, length);
    thrust::device_vector<In> inPtr               (1, in);
    thrust::device_vector<Out> outPtr             (1, out);

    if (verbose) cout << "in: " << in.epoch << " " << in.maxIterMult << endl;

    computePolysome <<< 1, numRibosomes >>> (thrust::raw_pointer_cast( codonsPtr.data() ), 
                                             thrust::raw_pointer_cast( ribosomesPtr.data() ), 
                                             thrust::raw_pointer_cast( lengthPtr.data() ), 
                                             thrust::raw_pointer_cast( inPtr.data() ),
                                             thrust::raw_pointer_cast( outPtr.data() ),
                                             deviceStates, verbose);

    out = outPtr[0];
    cout << "finished in " << out.iter << " iterations" << endl;
    if (out.iter >= in.maxIterMult * length)
        cerr << "warning: reached the maximum number of iterations" << endl;

    // debugging/visualization info
    if (verbose) printDebug (in, out, length);

    // write result
    probs.resize(length);
    hipMemcpy (&probs[0], deviceProb, length*sizeof(double), hipMemcpyDeviceToHost);

    // clean up
    hipFree (deviceProb);
    hipFree (deviceCodons);
    hipFree (deviceRibosomes);
    hipFree (deviceStates);
    
    gettimeofday(&tv2, NULL);
    printf("CUDA - main program: Time taken in execution = %f seconds\n",
           (double) (tv2.tv_usec - tv1.tv_usec) / (double)1000000 +
           (double) (tv2.tv_sec - tv1.tv_sec));
}



// sort input array and return permutation indices. Can be done with lambdas in Cuda 7.
// TODO: do sort in a kernel
struct LengthComparatorByIndex
{
    LengthComparatorByIndex (const vector<vector<double> >& data) : m_data(data) { }
    bool operator()(int left, int right) const { return m_data[left].size() < m_data[right].size(); }
    const vector< vector<double> > & m_data;
};
vector<size_t> orderedLength (const vector< vector<double> >& values) {
    vector<size_t> indices (values.size());
    for (int i = 0; i != values.size(); ++i) indices[i] = i;
    sort( indices.begin(), indices.end(), LengthComparatorByIndex(values));
    return indices;
}


void runMultiplePolysomes (const vector< vector<double> > rates, double epoch,
                           vector< vector<double> >& probs, int verbose)
{

    struct timeval tv1, tv2;
    
    gettimeofday(&tv1, NULL);
    hipFree(0);
    gettimeofday(&tv2, NULL);
    printf("CUDA init: Time taken in execution = %f seconds\n",
           (double) (tv2.tv_usec - tv1.tv_usec) / (double)1000000 +
           (double) (tv2.tv_sec - tv1.tv_sec));
           
    gettimeofday(&tv1, NULL);     
    int numRNAs = rates.size();
    probs.resize(numRNAs);

    const int MaxIterMult = 100;

    const int MinBlockPerSplit = 70;
    const double SplitReductionFactor = 1.5;
    // sort rates vectors based on length
    vector<size_t> indices = orderedLength (rates);
    // form a list of indices where RNAs will be split
    vector<int> indicesOfSplit (1, indices.size()-1);
    // split by length. Each time length is halved, it is a split
    for (int i = indices.size()-1; i != -1; --i)
    {
        // stop condition
        if (rates[indices[i]].size() < 32 * RiboWidth) break;
        // go until have at least 32 RNAs
        if (indicesOfSplit.back() - i < MinBlockPerSplit) continue;
        // point where one more split is done
        if (rates[indices[i]].size() < rates[indices[indicesOfSplit.back()]].size() / SplitReductionFactor)
            indicesOfSplit.push_back(i);
    }
    reverse (indicesOfSplit.begin(), indicesOfSplit.end());

    // info on the split
    cout << "size of splits: " << indicesOfSplit.size() << endl;
    for (int i = 0; i != indicesOfSplit.size(); ++i)
        cout << "split: " << setw(4) << rates[indices[indicesOfSplit[i]]].size() 
             << ", numRNA: " << indicesOfSplit[i] - (i == 0 ? 0 : indicesOfSplit[i-1]) << endl;
    cout << "end of splits." << endl;

    for (int split = 0; split != indicesOfSplit.size(); ++split)
    {
        int maxLength = rates[indices[indicesOfSplit[split]]].size();
        int numRibosomes = min (1024, ((maxLength / RiboWidth - 1) / 32 + 1) * 32);
        if (verbose > 1) cout << "maxLength: " << maxLength << ", numRibos: " << numRibosomes << endl;

        // set up seeds
        hiprandState* deviceStates;
        hipMalloc ( &deviceStates, numRibosomes * sizeof(hiprandState) );
        setupRand <<< 1, numRibosomes >>> ( deviceStates, time(NULL) );

        // one element per RNA
        thrust::device_vector<Codon*> codonsPtr       (numRNAs);
        thrust::device_vector<Ribosome*> ribosomesPtr (numRNAs);
        thrust::device_vector<int> lengthPtr          (numRNAs);
        thrust::device_vector<In> inPtr               (numRNAs);
        thrust::device_vector<Out> outPtr             (numRNAs);

        // prepare inputs
        int beginIndex = (split == 0 ? 0 : indicesOfSplit[split-1]) + 1;
        int endIndex = indicesOfSplit[split] + 1;
        for (int index = beginIndex; index != endIndex; ++index)
        {
            int rna = indices[index];
            int length = rates[rna].size();

            // init
            Ribosome* deviceRibosomes = initRibosomes(numRibosomes);
            Codon*    deviceCodons    = initCodons(rates[rna]);

            // pass constants
            In in; in.epoch = epoch; in.maxIterMult = MaxIterMult; in.frontpadding = 1;

            // info to return
            double* deviceProb;
            hipMalloc (&deviceProb, length*sizeof(double));
            Out out; out.prob = deviceProb;

            // debugging/visualization info to return
            if (verbose > 1) initDebug (in, out, length, (verbose > 1 ? 200 : 0));

            // copy the in/out structs to device
            codonsPtr[rna]    = deviceCodons;
            ribosomesPtr[rna] = deviceRibosomes;
            lengthPtr[rna]    = length;
            inPtr[rna]        = in;
            outPtr[rna]       = out;
        }

        computePolysome <<< numRNAs, numRibosomes >>> (thrust::raw_pointer_cast( codonsPtr.data() ), 
                                                       thrust::raw_pointer_cast( ribosomesPtr.data() ), 
                                                       thrust::raw_pointer_cast( lengthPtr.data() ), 
                                                       thrust::raw_pointer_cast( inPtr.data() ),
                                                       thrust::raw_pointer_cast( outPtr.data() ),
                                                       deviceStates, verbose);

        // process outputs
        for (int index = beginIndex; index != endIndex; ++index)
        {
            int rna = indices[index];
            int length = rates[rna].size();

            Codon* deviceCodons = codonsPtr[rna];
            Ribosome* deviceRibosomes = ribosomesPtr[rna];
            Out out = outPtr[rna];
            In in = inPtr[rna];
            double* deviceProb = out.prob;

            int numRibosomes = min (1024, ((maxLength / RiboWidth - 1) / 32 + 1) * 32);
            if (verbose)
                cout << "rna: " << setw(4) << rna 
                     << ", length: " << setw(4) << length 
                     << ", numRibos: " << setw(4) << numRibosomes
                     << ", finished in " << out.iter << " iterations" << endl;
            if (out.iter >= in.maxIterMult * length)
                cerr << "warning: reached the maximum number of iterations" << endl;

            // debugging/visualization info
            if (verbose > 1) printDebug (in, out, length);

            // write result
            probs[rna].resize(length);
            hipMemcpy (&probs[rna][0], deviceProb, length*sizeof(double), hipMemcpyDeviceToHost);

            // clean up
            hipFree (deviceProb); 
            hipFree (deviceCodons);
            hipFree (deviceRibosomes);
        }

        hipFree (deviceStates);
    } // split
    gettimeofday(&tv2, NULL);
    printf("CUDA main: Time taken in execution = %f seconds\n",
           (double) (tv2.tv_usec - tv1.tv_usec) / (double)1000000 +
           (double) (tv2.tv_sec - tv1.tv_sec));
}
