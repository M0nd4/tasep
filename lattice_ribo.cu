#include "hip/hip_runtime.h"
#include "lattice.hpp"

#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <thrust/device_vector.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iomanip>

__global__ static 
void setupRand ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
} 


__device__ inline static
double sampleTime (double rate, hiprandState* globalState)
{
    hiprandState localState = globalState[threadIdx.x];
    double u = hiprand_uniform_double( &localState );
    globalState[threadIdx.x] = localState;
    if (u == 0) u += 0.1; // should almost never happen
    return -log(u) / rate;  // let rate == 0 throw
}


__device__ static
//__global__ static 
void updatePolysome (Codon* codons, Ribosome* ribosomes, int length, double epoch, hiprandState* globalState)
{
    // TODO: ribosomes AoS to SoA

    // copy ribosom data from global memory to registers
    int riboId = threadIdx.x;
    int nextId = (riboId + blockDim.x - 1) % blockDim.x;
    Ribosome ribo     = ribosomes[riboId];
    Ribosome nextribo = ribosomes[nextId];

    // TODO: many ribosomes will be in pos == 0. Check early

    // copy codon data from global memory to registers
    int pos = ribo.pos;
    int nextpos = (pos + length + 1) % length;
    Codon codon = codons[pos];
    Codon nextcodon = codons[nextpos];
    __syncthreads();

    // update current time with time of the next codon
    double t0 = max(codon.time, nextcodon.time);
    ribosomes[riboId].time = t0;
    codons[pos].accumtime += t0 - codon.time; 
    codons[pos].time = t0;
    
    // do not jump if can not
    bool nextIsFar = (nextribo.pos - pos + length) % length > 1 || (pos != 0 && nextribo.pos == 0); 
    if (!nextIsFar) return;

   // sample the period
    double dt = sampleTime(codon.rate, globalState);
    dt = min(dt, epoch - t0);     // when about to finish
    double t = t0 + dt;

    // update times of the ribo and of the codon
    codons[pos].accumtime += dt;
    ribosomes[riboId].time = t;
    codons[pos].time = t;         // for the next ribosome
    codons[nextpos].time = t;     // for computing time of occupancy by next ribosome

    // finish simulation for this ribo when time reaches the epoch
    if (t >= epoch) return;

    // flip two codons and update the ribosome position
    codons[nextpos].occupied = true;
    codons[pos].occupied = false;
    ribosomes[riboId].pos = nextpos;

     // take care of the border
    if (pos == length - 1)
    {
        ribosomes[riboId].time = 0;
        codons[0].time = 0;
        codons[0].occupied = false;
    }
}


__device__ static
bool stopCondition (Ribosome* ribosomes, double epoch)
{
    // TODO: rewrite with reduce
        int countNonactive = 0;
        for (int i = 0; i != blockDim.x; ++i)
        {
            Ribosome ribo = ribosomes[i];
            if (ribo.time >= epoch || ribo.pos == 0)
                ++countNonactive;
        }
        int countActive = blockDim.x - countNonactive;
        return (countActive == 0);
}


__global__ static 
void computePolysome (Codon** codonsPtr, Ribosome** ribosomesPtr, int* lengthPtr, 
                      double epoch, hiprandState* globalState)
{
    __shared__ bool flag_terminate;
   
    // each block has its own arrays (numRibosomes is the same in every block)
    Codon*    codons = codonsPtr[blockIdx.x];
    Ribosome* ribosomes = ribosomesPtr[blockIdx.x];
    int       length = lengthPtr[blockIdx.x];

    const int MaxIter = 1000 * length;
    for (int it = 0; it != MaxIter; ++it)
    {
        //if (threadIdx.x == 0)
        //    flag_terminate = it > 2;// stopCondition(ribosomes, epoch);
        //__syncthreads();
        //if (flag_terminate) break;

        return;

        /*
        if (verbose)
        {
        cout << setw(2) <<  it << "  &  " << countActive << "  &   " << flush;
        for (int i = 0; i != lengthPadded; ++i)
        {
            Codon codon = codonsVector[i];
            cout << (codon.occupied ? '*' : '.');
        }
        cout << "  &  ";
        for (int i = padding; i != lengthPadded; ++i)
        {
            Codon codon = codonsVector[i];
            cout << setprecision(2) << setw(2) << codon.accumtime << " ";
        }
        cout << " \\\\" << endl;
        }
        */

        updatePolysome (codons, ribosomes, length, epoch, globalState);
    }

    /*
    cout << "finished in " << it << " iterations" << endl;
    if (it == MaxIter)
        cerr << "warning: reached the maximum number of iterations" << endl;
    */


}


using namespace std;


vector<double> runSinglePolysome (const vector<double>& rates, double initRate, double epoch, int verbose)
{
    int padding = 1;

    // pad the vector
    int length = rates.size();
    int lengthPadded = length + padding;

    cout << "length: " << length << endl;
    cout << "padding: " << padding << endl;

    // init codons
    thrust::device_vector<Codon> codonsVector (lengthPadded);
    for (int i = 0; i != length; ++i)
    {
        Codon codon; codon.rate = rates[i]; codon.time = 0; codon.occupied = false; codon.accumtime = 0;
        codonsVector[i+padding] = codon;
    }
    Codon codon0; codon0.rate = initRate; codon0.time = 0; codon0.occupied = false; codon0.accumtime = 0;
    codonsVector.front() = codon0;
    Codon codon1 = codonsVector[1];
    codon1.occupied = true;
    codonsVector[1] = codon1;
    Codon* deviceCodons = thrust::raw_pointer_cast( &codonsVector[0] );

    // init ribosomes
    //const int RiboWidth = 10;
    const int numRibosomes = lengthPadded;//((lengthPadded - 1) / 32 / RiboWidth + 1) * 32;
    cout << "numRibosomes: " << numRibosomes << endl;
    thrust::device_vector<Ribosome> ribosomesVector (numRibosomes);
    Ribosome ribo00; ribo00.pos = 0; ribo00.time = 0;
    Ribosome ribo10; ribo10.pos = 1; ribo10.time = 0;
    for (int i = 0; i != numRibosomes; ++i) ribosomesVector[i] = ribo00;
    ribosomesVector[0] = ribo10;
    Ribosome* deviceRibosomes = thrust::raw_pointer_cast( &ribosomesVector[0] );

    // set up seeds
    hiprandState* deviceStates;
    hipMalloc ( &deviceStates, numRibosomes * sizeof(hiprandState) );
    setupRand <<< 1, numRibosomes >>> ( deviceStates, time(NULL) );

    // store (in a single case, a single one) pointers to codons and ribosomes by block.
    // just need to have a pair of pointer in global memory
    thrust::device_ptr< Codon* >    codonsPtr (&deviceCodons);
    thrust::device_ptr< Ribosome* > ribosomesPtr (&deviceRibosomes);
    thrust::device_ptr< int >       lengthPtr (&lengthPadded);

    computePolysome <<< 1, numRibosomes >>> (thrust::raw_pointer_cast( codonsPtr ), 
                                             thrust::raw_pointer_cast( ribosomesPtr ), 
                                             thrust::raw_pointer_cast( lengthPtr ), 
                                             epoch, deviceStates);

    vector<double> probs (length);
    /*for (int i = 0; i != probs.size(); ++i)
    {
        Codon codon = codonsVector[i+padding];
        probs[i] = codon.accumtime / epoch;
    }
    */

    return probs;
}
