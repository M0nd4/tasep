#include "hip/hip_runtime.h"
#include "lattice.hpp"

#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <algorithm>

#include <thrust/device_vector.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iomanip>

// FIXME: for many blocks probably need to create many states
__global__ static
void setupRand ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
} 


__device__ inline static
double sampleTime (double rate, hiprandState* globalState)
{
    hiprandState localState = globalState[threadIdx.x];
    double u = hiprand_uniform_double( &localState );
    globalState[threadIdx.x] = localState;
    if (u == 0) u += 0.1; // should almost never happen
    return -log(u) / rate;  // let rate == 0 throw
}


__device__ static
//__global__ static 
void updatePolysome (Codon* codons, Ribosome* ribosomes, int length, double epoch, hiprandState* globalState)
{
    // TODO: ribosomes AoS to SoA

    // copy ribosom data from global memory to registers
    int riboId = threadIdx.x;
    int nextId = (riboId + blockDim.x - 1) % blockDim.x;
    Ribosome ribo     = ribosomes[riboId];
    Ribosome nextribo = ribosomes[nextId];

    // TODO: many ribosomes will be in pos == 0. Check early

    // copy codon data from global memory to registers
    int pos = ribo.pos;
    int nextpos = (pos + length + 1) % length;
    Codon codon = codons[pos];
    Codon nextcodon = codons[nextpos];

    // after copy
    __syncthreads();

    // update current time with time of the next codon
    double t0 = max(codon.time, nextcodon.time);
    ribosomes[riboId].time = t0;
    codons[pos].accumtime += t0 - codon.time; 
    codons[pos].time = t0;
    
    // do not jump if can not
    bool nextIsFar = (nextribo.pos - pos + length) % length > 1 || (pos != 0 && nextribo.pos == 0); 
    if (!nextIsFar) return;

   // sample the period
    double dt = sampleTime(codon.rate, globalState);
    dt = min(dt, epoch - t0);     // when about to finish
    double t = t0 + dt;

    // update times of the ribo and of the codon
    codons[pos].accumtime += dt;
    ribosomes[riboId].time = t;
    codons[pos].time = t;         // for the next ribosome
    codons[nextpos].time = t;     // for computing time of occupancy by next ribosome

    // finish simulation for this ribo when time reaches the epoch
    if (t >= epoch) return;

    // flip two codons and update the ribosome position
    codons[nextpos].occupied = true;
    codons[pos].occupied = false;
    ribosomes[riboId].pos = nextpos;

     // take care of the border
    if (pos == length - 1)
    {
        ribosomes[riboId].time = 0;
        codons[0].time = 0;
        codons[0].occupied = false;
    }
}


__device__ static
int countActiveRibos (Ribosome* ribosomes, double epoch)
{
    // TODO: rewrite with reduce
    int countNonactive = 0;
    for (int i = 0; i != blockDim.x; ++i)
    {
        Ribosome ribo = ribosomes[i];
        if (ribo.time >= epoch || ribo.pos == 0)
            ++countNonactive;
    }
    return blockDim.x - countNonactive;
}


// pass info forward and backwards
struct In {
    int maxIterMult;
    double epoch;
    int frontpadding;
    int iters4display;
};

struct Out {
    int iter;
    double* prob;
    char* occupancy;
    int* activeRibos;
};


__global__ static 
void computePolysome (Codon** codonsPtr, Ribosome** ribosomesPtr, int* lengthsPtr, 
                      In* inPtr, Out* outPtr, hiprandState* globalState)
{
    __shared__ int activeRibos;
   
    // each block has its own arrays (numRibosomes is the same in every block)
    Codon*    codons = codonsPtr[blockIdx.x];
    Ribosome* ribosomes = ribosomesPtr[blockIdx.x];
    int       length = lengthsPtr[blockIdx.x];
    In        in = inPtr[blockIdx.x];
    Out       out = outPtr[blockIdx.x];

    for (out.iter = 0; out.iter != in.maxIterMult * length; ++out.iter)
    {
        // stop condition
        if (threadIdx.x == 0)
            activeRibos = countActiveRibos (ribosomes, in.epoch);
        __syncthreads();
        if (activeRibos == 0) break;

        // write occupancy into specially pre-allocated memory 
        if (out.iter < in.iters4display)
        {
            if (threadIdx.x == 0) out.activeRibos[out.iter] = activeRibos;
            for (int i = threadIdx.x; i < length; i += blockDim.x)
                out.occupancy[out.iter * length + i] = codons[i].occupied;
            __syncthreads();
        }

        updatePolysome (codons, ribosomes, length, in.epoch, globalState);
    }

    // calculate resulting probability
    __syncthreads();
    
    for (int i = threadIdx.x; i < length; i += blockDim.x)
        out.prob[i] = codons[i+in.frontpadding].accumtime / in.epoch;
    
    if (threadIdx.x == 0)
        outPtr[blockIdx.x] = out;
}



using namespace std;


void runSinglePolysome (const vector<double>& rates, double epoch, 
                        vector<double>& probs, int verbose)
{
    int padding = 1;

    // pad the vector
    int lengthPadded = rates.size();
    int length = rates.size() - 1;  // first element is initRate

    cout << "length: " << length << endl;
    cout << "padding: " << padding << endl;

    // init codons
    // TODO: in a kernel
    thrust::device_vector<Codon> codonsVector (lengthPadded);
    for (int i = 0; i != lengthPadded; ++i)
    {
        Codon codon; codon.rate = rates[i]; codon.time = 0; codon.occupied = false; codon.accumtime = 0;
        codonsVector[i] = codon;
    }
    Codon codon1 = codonsVector[1]; codon1.occupied = true; codonsVector[1] = codon1;
    Codon* deviceCodons = thrust::raw_pointer_cast( &codonsVector[0] );

    // init ribosomes
    // TODO: in a kernel
    const int numRibosomes = ((lengthPadded - 1) / 32 + 1) * 32;
    cout << "numRibosomes: " << numRibosomes << endl;
    thrust::device_vector<Ribosome> ribosomesVector (numRibosomes);
    Ribosome ribo00; ribo00.pos = 0; ribo00.time = 0;
    Ribosome ribo10; ribo10.pos = 1; ribo10.time = 0;
    for (int i = 0; i != numRibosomes; ++i) ribosomesVector[i] = ribo00;
    ribosomesVector[0] = ribo10;
    Ribosome* deviceRibosomes = thrust::raw_pointer_cast( &ribosomesVector[0] );

    // set up seeds
    hiprandState* deviceStates;
    hipMalloc ( &deviceStates, numRibosomes * sizeof(hiprandState) );
    setupRand <<< 1, numRibosomes >>> ( deviceStates, time(NULL) );

    // it's easy to copy to vector, so let's have vectors of length 1
    thrust::device_vector<Codon*> codonsPtr       (1, deviceCodons);
    thrust::device_vector<Ribosome*> ribosomesPtr (1, deviceRibosomes);
    thrust::device_vector<int> lengthPtr          (1, lengthPadded);

    // info to give
    In in; in.epoch = epoch; in.maxIterMult = 1000; in.frontpadding = 1;

    // info to return
    double* deviceProb;
    hipMalloc (&deviceProb, lengthPadded*sizeof(double));
    Out out; out.prob = deviceProb;

    // debugging/visualization info to return
    int iters4display = verbose >= 2 ? 200 : 0;
    int space4occupancy = iters4display*lengthPadded*sizeof(char);
    char* deviceOccupancy;
    int* deviceActiveRibos;
    if (iters4display) 
    {
        hipMalloc (&deviceOccupancy, space4occupancy);
        hipMemset (deviceOccupancy, 0, space4occupancy);
        hipMalloc (&deviceActiveRibos, iters4display*sizeof(int));
        hipMemset (deviceActiveRibos, 0, iters4display*sizeof(int));
    }
    in.iters4display = iters4display;
    out.occupancy = deviceOccupancy;
    out.activeRibos = deviceActiveRibos;
                            
    // copy the in/out structs to device
    thrust::device_vector<In> inPtr               (1, in);
    thrust::device_vector<Out> outPtr             (1, out);

    if (verbose)
        cout << "in: " << in.epoch << " " << in.maxIterMult << endl;

    computePolysome <<< 1, numRibosomes >>> (thrust::raw_pointer_cast( &codonsPtr[0] ), 
                                             thrust::raw_pointer_cast( &ribosomesPtr[0] ), 
                                             thrust::raw_pointer_cast( &lengthPtr[0] ), 
                                             thrust::raw_pointer_cast( &inPtr[0] ),
                                             thrust::raw_pointer_cast( &outPtr[0] ),
                                             deviceStates);

    out = outPtr[0];
    cout << "finished in " << out.iter << " iterations" << endl;
    if (out.iter >= in.maxIterMult * length)
        cerr << "warning: reached the maximum number of iterations" << endl;

    // debugging/visualization info
    if (iters4display)
    {
        vector<char> vectorOccupancy (iters4display*lengthPadded);
        hipMemcpy (&vectorOccupancy[0], deviceOccupancy, space4occupancy, hipMemcpyDeviceToHost);
        vector<int> vectorActiveRibos (iters4display);
        hipMemcpy (&vectorActiveRibos[0], deviceActiveRibos, iters4display*sizeof(int), hipMemcpyDeviceToHost);
        for (int iter = 0; iter != min(iters4display, out.iter); ++iter)
        {
            cout << setw(3) << iter << "  &  " << setw(3) << vectorActiveRibos[iter] << "  &  ";
            for (int i = 0; i != lengthPadded; ++i)
                cout << (vectorOccupancy[iter * lengthPadded + i] ? '*' : '.');
            cout << endl;
        }
        hipFree (deviceOccupancy);
        hipFree (deviceActiveRibos);
    }
     
    // write result
    probs.resize(length);
    hipMemcpy (&probs[0], deviceProb, length*sizeof(double), hipMemcpyDeviceToHost);
    hipFree (deviceProb);
}



// sort input array and return permutation indices. Can be done with lambdas in Cuda 7.
// TODO: do sort in a kernel
/*
struct LengthComparatorByIndex
{
    LengthComparatorByIndex (const vector<vector<double> >& data) : m_data(data) { }
    bool operator()(int left, int right) const { return m_data[left].size() < m_data[right].size(); }
    const vector< vector<double> > & m_data;
};
vector<size_t> orderedLength (const vector< vector<double> >& values) {
    vector<size_t> indices (values.size());
    for (int i = 0; i != values.size(); ++i) indices[i] = i;
    sort( indices.begin(), indices.end(), LengthComparatorByIndex(values));
    return indices;
}
*/

bool lengthCompare (const vector<double>& left, const vector<double>& right)
{
    return left.size() < right.size();
}

void runMultiplePolysomes (const vector< vector<double> > rates, double epoch,
                           vector< vector<double> >& probs, int verbose)
{
    int numRNAs = rates.size();
    probs.resize(numRNAs);

    // sort rates vectors based on length
    //vector<size_t> indices = orderedLength (rates);

    const int MaxIterMult = 1000;
    const int iters4display = verbose >= 2 ? 200 : 0;
    
    int maxLengthPadded = max_element(rates.begin(), rates.end(), lengthCompare)->size();
    int numRibosomes = ((maxLengthPadded - 1) / 32 + 1) * 32;
    if (verbose)
        cout << "epoch: " << epoch << ", MaxIterMult: " << MaxIterMult << ", numRibos: " << numRibosomes << endl;

    // set up seeds
    hiprandState* deviceStates;
    hipMalloc ( &deviceStates, numRibosomes * sizeof(hiprandState) );
    setupRand <<< 1, numRibosomes >>> ( deviceStates, time(NULL) );

    // one element per RNA
    thrust::device_vector<Codon*> codonsPtr       (numRNAs);
    thrust::device_vector<Ribosome*> ribosomesPtr (numRNAs);
    thrust::device_vector<int> lengthPtr          (numRNAs);
    thrust::device_vector<In> inPtr               (numRNAs);
    thrust::device_vector<Out> outPtr             (numRNAs);

    // prepare inputs
    for (int rna = 0; rna != numRNAs; ++rna)
    {
        int lengthPadded = rates[rna].size();

        // init codons
        // TODO: in a kernel
        thrust::device_vector<Codon> codonsVector (lengthPadded);
        for (int i = 0; i != lengthPadded; ++i)
        {
            Codon codon; codon.rate = rates[rna][i]; codon.time = 0; codon.occupied = false; codon.accumtime = 0;
            codonsVector[i] = codon;
        }
        Codon codon1 = codonsVector[1]; codon1.occupied = true; codonsVector[1] = codon1;
        Codon* deviceCodons = thrust::raw_pointer_cast( &codonsVector[0] );

        // init ribosomes
        // TODO: in a kernel
        thrust::device_vector<Ribosome> ribosomesVector (numRibosomes);
        Ribosome ribo00; ribo00.pos = 0; ribo00.time = 0;
        Ribosome ribo10; ribo10.pos = 1; ribo10.time = 0;
        for (int i = 0; i != numRibosomes; ++i) ribosomesVector[i] = ribo00;
        ribosomesVector[0] = ribo10;
        Ribosome* deviceRibosomes = thrust::raw_pointer_cast( &ribosomesVector[0] );

        // info to give
        In in; in.epoch = epoch; in.maxIterMult = 1000; in.frontpadding = 1;
        //in.iters4display = 0;

        // info to return
        double* deviceProb;
        hipMalloc (&deviceProb, lengthPadded*sizeof(double));
        Out out; out.prob = deviceProb;

        // debugging/visualization info to return
        int space4occupancy = iters4display*lengthPadded*sizeof(char);
        char* deviceOccupancy;
        int* deviceActiveRibos;
        if (iters4display) 
        {
            hipMalloc (&deviceOccupancy, space4occupancy);
            hipMemset (deviceOccupancy, 0, space4occupancy);
            hipMalloc (&deviceActiveRibos, iters4display*sizeof(int));
            hipMemset (deviceActiveRibos, 0, iters4display*sizeof(int));
        }
        in.iters4display = iters4display;
        out.occupancy = deviceOccupancy;
        out.activeRibos = deviceActiveRibos;

        // copy the in/out structs to device
        codonsPtr[rna] = deviceCodons;
        ribosomesPtr[rna] = deviceRibosomes;
        lengthPtr[rna] = lengthPadded;
        inPtr[rna] = in;
        outPtr[rna] = out;

        if (verbose > 1)
            cout << "rna: " << rna << ", length: " << lengthPadded-1 << endl;
    }

    computePolysome <<< numRNAs, numRibosomes >>> (thrust::raw_pointer_cast( &codonsPtr[0] ), 
                                                   thrust::raw_pointer_cast( &ribosomesPtr[0] ), 
                                                   thrust::raw_pointer_cast( &lengthPtr[0] ), 
                                                   thrust::raw_pointer_cast( &inPtr[0] ),
                                                   thrust::raw_pointer_cast( &outPtr[0] ),
                                                   deviceStates);

    // process outputs
    for (int rna = 0; rna != numRNAs; ++rna)
    {
        int lengthPadded = rates[rna].size();
        int length = lengthPadded - 1;
        Out out = outPtr[rna];
        In in = inPtr[rna];
        double* deviceProb = out.prob;
        char* deviceOccupancy = out.occupancy;
        int* deviceActiveRibos = out.activeRibos;

        if (verbose)
            cout << "rna: " << rna << ", finished in " << out.iter << " iterations" << endl;
        if (out.iter >= in.maxIterMult * length)
            cerr << "warning: reached the maximum number of iterations" << endl;

        // debugging/visualization info
        if (iters4display)
        {
            int space4occupancy = iters4display*lengthPadded*sizeof(char);
            vector<char> vectorOccupancy (iters4display*lengthPadded);
            hipMemcpy (&vectorOccupancy[0], deviceOccupancy, space4occupancy, hipMemcpyDeviceToHost);
            vector<int> vectorActiveRibos (iters4display);
            hipMemcpy (&vectorActiveRibos[0], deviceActiveRibos, iters4display*sizeof(int), hipMemcpyDeviceToHost);
            for (int iter = 0; iter != min(iters4display, out.iter); ++iter)
            {
                cout << setw(3) << iter << "  &  " << setw(3) << vectorActiveRibos[iter] << "  &  ";
                for (int i = 0; i != lengthPadded; ++i)
                    cout << (vectorOccupancy[iter * lengthPadded + i] ? '*' : '.');
                cout << endl;
            }
            hipFree (deviceOccupancy);
            hipFree (deviceActiveRibos);
        }
         
        // write result
        probs[rna].resize(length);
        hipMemcpy (&probs[rna][0], deviceProb, length*sizeof(double), hipMemcpyDeviceToHost);
        hipFree (deviceProb); 
    }
}

